/*
	Object:		Raster-scan and label-equivalence-based algorithm.
	Authors:	Giuliano Langella & Massimo Nicolazzo
	email:		gyuliano@libero.it


-----------
DESCRIPTION:
-----------

 I: "urban"		--> [0,0] shifted
 O: "lab_mat"	--> [1,1] shifted

	The "forward scan mask" for eight connected connectivity is the following:
		nw		nn		ne
		ww		cc		xx
		xx		xx		xx
	assuming that:
		> cc is the background(=0)/foreground(=1) pixel at (r,c),
		> nw, nn, ne, ww are the north-west, north, north-east and west pixels in the eight connected connectivity,
		> xx are skipped pixels.
	Therefore the mask has 4 active pixels with(out) object pixels (that is foreground pixels).

*/

//	INCLUDES
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <errno.h>        	/* errno */
#include <string.h>       	/* strerror */
#include <math.h>			// ceil
#include <time.h>			// CLOCKS_PER_SEC
//#include </usr/local/cuda/samples/common/inc/helper_cuda.h>	// helper for checking cuda initialization and error checking
//#include </usr/local/cuda/samples/common/inc/helper_string.h>

// CUDA
#include <hip/hip_runtime.h>


//	-indexes
#define durban(cc,rr)	urban[		(cc)	+	(rr)	*(blockDim.x)	] // I: scan value at current [r,c]
#define nw_pol(cc,rr)	lab_mat_sh[	(cc-1)	+	(rr-1)	*(blockDim.x)	] // O: scan value at North-West
#define nn_pol(cc,rr)	lab_mat_sh[	(cc+0)	+	(rr-1)	*(blockDim.x)	] // O: scan value at North
#define ne_pol(cc,rr)	lab_mat_sh[	(cc+1)	+	(rr-1)	*(blockDim.x)	] // O: scan value at North-East
#define ww_pol(cc,rr)	lab_mat_sh[	(cc-1)	+	(rr+0)	*(blockDim.x)	] // O: scan value at West
#define ee_pol(cc,rr)	lab_mat_sh[	(cc+1)	+	(rr+0)	*(blockDim.x)	] // O: scan value at West
#define sw_pol(cc,rr)	lab_mat_sh[	(cc-1)	+	(rr+1)	*(blockDim.x)	] // O: scan value at South-West
#define ss_pol(cc,rr)	lab_mat_sh[	(cc+0)	+	(rr+1)	*(blockDim.x)	] // O: scan value at South-West
#define se_pol(cc,rr)	lab_mat_sh[	(cc+1)	+	(rr+1)	*(blockDim.x)	] // O: scan value at South-West
#define cc_pol(cc,rr)	lab_mat_sh[	(cc+0)	+	(rr+0)	*(blockDim.x)	] // O: scan value at current [r,c] which is shifted by [1,1] in O

// GLOBAL VARIABLES
#define			Vo			1	// object value
#define			Vb			0	// object value
char			buffer[255];
const char		*Lcuda		= "/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/CUDA-code.txt";
const char 		*ALL_txt	= "/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/ALL.txt";
/*
 * 	To test my CCL .cu code, I can use the following input parameters:
 * 		8 8 6999 6999
 * 	and NTHREADSX =
 * 		8
 * 	with a very large file called:
 * 		const char 		*ALL_txt	= "/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/ALL-big-test.txt";
 */

//
/*
unsigned int r 			= threadIdx.y;
unsigned int c 			= threadIdx.x;
unsigned int bdx		= blockDim.x;
unsigned int bdy		= blockDim.y;
unsigned int bix		= blockIdx.x;
unsigned int biy		= blockIdx.y;
unsigned int gdx		= gridDim.x;
unsigned int gdy		= gridDim.y;
unsigned int iTile		= gdx * biy + bix;
*/
/*
#define	 		r 		threadIdx.y
#define			c		threadIdx.x
#define			bdx		blockDim.x
#define			bdy		blockDim.y
#define			bix		blockIdx.x
#define			biy		blockIdx.y
#define			gdx		gridDim.x
#define			gdy		gridDim.y
#define			iTile	gdx * biy + bix
*/

//---------------------------- FUNCTIONS PROTOTYPES
//		** I/O **
void read_urbmat(unsigned char *, unsigned int, unsigned int, const char *);
void write_urbmat_tiled( unsigned char *, unsigned int, unsigned int, unsigned int, unsigned int, const char *);
void write_urbmat_matlab( unsigned char *, unsigned int, unsigned int, unsigned int, unsigned int, const char *);
void write_labmat_tiled( unsigned int *,  unsigned int, unsigned int, unsigned int, unsigned int, const char *);
void write_labmat_matlab(unsigned int *,  unsigned int, unsigned int, unsigned int, unsigned int, const char *);
//		** kernels **
//	(1)
__global__ void intra_tile_labeling( const unsigned char *,unsigned int, unsigned int * );
//	(2)
__global__ void stitching_tiles( unsigned int *,const unsigned int,const unsigned int );
//	(3)
__global__ void root_equivalence( unsigned int *,const unsigned int,const unsigned int );
//	(4)
__global__ void intra_tile_re_label(unsigned int,unsigned int *);

//		** OLD kernels **
__global__ void inter_tile_labeling( unsigned int *, bool );
//---------------------------- FUNCTIONS PROTOTYPES

void read_urbmat(unsigned char *urban, unsigned int nrows, unsigned int ncols, const char *filename)
{
	/*
	 * 	This function reads the Image and store in RAM with a 1-pixel-width zero-padding.
	 */
	unsigned int rr,cc;
	FILE *fid ;
	int a;
	fid = fopen(filename,"rt");
	if (fid == NULL) { printf("Error opening file:\n\t%s\n",filename); exit(1); }
	for(rr=0;rr<nrows;rr++) for(cc=0;cc<ncols;cc++) urban[cc+rr*ncols] = 0;
	for(rr=1;rr<nrows-1;rr++){
		for(cc=1;cc<ncols-1;cc++){
			fscanf(fid, "%d",&a);
			urban[cc+rr*ncols]=(unsigned char)a;
			//printf("%d ",a);
		}
		//printf("\n");
	}
	fclose(fid);
}
void write_urbmat_tiled(unsigned char *urb_mat, unsigned int nr, unsigned int nc, unsigned int ntilesX, unsigned int ntilesY, const char *filename)
{
	unsigned long int rr,cc,ntX,ntY;
	FILE *fid ;
	fid = fopen(filename,"w");
	if (fid == NULL) { printf("Error opening file %s!\n",filename); exit(1); }
	long long int offset;

	for(ntY=0;ntY<ntilesY;ntY++)
	{
		for(rr=0;rr<nr;rr++)
		{
			for(ntX=0;ntX<ntilesX;ntX++)
			{
				for(cc=0;cc<nc;cc++)
				{
					/*if( !(((cc==nc-1) && ((ntilesX*ntY+ntX+1)%ntilesX)==0))	&&	// do not print last column
						!(((rr==nr-1) && (ntY==ntilesY-1))) 					// do not print last row
					)*/
					{
						offset = (ntilesX*ntY+ntX)*nc*nr+(nc*rr+cc);
						fprintf(fid, "%6d ",urb_mat[offset]);
						//printf(		 "%d ",lab_mat[offset]);
					}
				}
				fprintf(fid,"\t\t");
				//printf(		"\n");
			}
			fprintf(fid,"\n");
			//printf(		"\n");
		}
		fprintf(fid,"\n");
	}
	fclose(fid);
}
void write_urbmat_matlab(unsigned char *urb_mat, unsigned int nr, unsigned int nc, unsigned int ntilesX, unsigned int ntilesY, const char *filename)
{
	unsigned int rr,cc,ntX,ntY;
	FILE *fid ;
	fid = fopen(filename,"w");
	if (fid == NULL) { printf("Error opening file %s!\n",filename); exit(1); }
	int offset;

	for(ntY=0;ntY<ntilesY;ntY++)
	{
		for(rr=1;rr<nr;rr++)
		{
			for(ntX=0;ntX<ntilesX;ntX++)
			{
				for(cc=1;cc<nc;cc++)
				{
					if( !(((cc==nc-1) && ((ntilesX*ntY+ntX+1)%ntilesX)==0))	&&	// do not print last column
						!(((rr==nr-1) && (ntY==ntilesY-1))) 					// do not print last row

					)
					{
						offset = (ntilesX*ntY+ntX)*nc*nr+(nc*rr+cc);
						fprintf(fid, "%6d ",urb_mat[offset]);
						//printf(		 "%d ",lab_mat[offset]);
					}
				}
			}
			fprintf(fid,"\n");
			//printf(		"\n");
		}
	}
	fclose(fid);
}
void write_labmat_tiled(unsigned int *lab_mat, unsigned int nr, unsigned int nc, unsigned int ntilesX, unsigned int ntilesY, const char *filename)
{
	unsigned int rr,cc,ntX,ntY;
	FILE *fid ;
	fid = fopen(filename,"w");
	if (fid == NULL) { printf("Error opening file %s!\n",filename); exit(1); }
	int offset;

	for(ntY=0;ntY<ntilesY;ntY++)
	{
		for(rr=0;rr<nr;rr++)
		{
			for(ntX=0;ntX<ntilesX;ntX++)
			{
				for(cc=0;cc<nc;cc++)
				{
					/*if( !(((cc==nc-1) && ((ntilesX*ntY+ntX+1)%ntilesX)==0))	&&	// do not print last column
						!(((rr==nr-1) && (ntY==ntilesY-1))) 					// do not print last row
					)*/
					{
						offset = (ntilesX*ntY+ntX)*nc*nr+(nc*rr+cc);
						fprintf(fid, "%6d ",lab_mat[offset]);
						//printf(		 "%d ",lab_mat[offset]);
					}
				}
				fprintf(fid,"\t\t");
				//printf(		"\n");
			}
			fprintf(fid,"\n");
			//printf(		"\n");
		}
		fprintf(fid,"\n");
	}
	fclose(fid);
}
void write_labmat_matlab(unsigned int *lab_mat, unsigned int nr, unsigned int nc, unsigned int ntilesX, unsigned int ntilesY, const char *filename)
{
	unsigned int rr,cc,ntX,ntY;
	FILE *fid ;
	fid = fopen(filename,"w");
	if (fid == NULL) { printf("Error opening file %s!\n",filename); exit(1); }
	int offset;

	for(ntY=0;ntY<ntilesY;ntY++)
	{
		for(rr=1;rr<nr;rr++)
		{
			for(ntX=0;ntX<ntilesX;ntX++)
			{
				for(cc=1;cc<nc;cc++)
				{
					if( !(((cc==nc-1) && ((ntilesX*ntY+ntX+1)%ntilesX)==0))	&&	// do not print last column
						!(((rr==nr-1) && (ntY==ntilesY-1))) 					// do not print last row

					)
					{
						offset = (ntilesX*ntY+ntX)*nc*nr+(nc*rr+cc);
						fprintf(fid, "%6d ",lab_mat[offset]);
						//printf(		 "%d ",lab_mat[offset]);
					}
				}
			}
			fprintf(fid,"\n");
			//printf(		"\n");
		}
	}
	fclose(fid);
}
__global__ void inter_tile_labeling( unsigned int *lm, bool *gfound )
{
	/*
	 * IMPORTANT NOTE:
	 * 	> 	We only need to read the shared border vector from {nn,ww,ee,ss} tiles and not the entire adjacent tiles!
	 * 		Hence we should modify the code accordingly!
	 * 	>	We should allocate using cudaMallocPitch and not cudaMalloc: nvidia says that it is faster!!
	 * 		See CUDA_Runtime_API.pdf, page 92.
	 * 	>	Invertire i due if nei blocchi << if(XX_tile>=0) >> in modo che il secondo if
	 * 		(ossia accendere i thread in cc con ID uguale a quello in indice ii del bordo)
	 * 		venga eseguito solo se il primo if
	 * 		(ossia che il valore sul bordo della tile {nn,ww,ee,ss} adiacente è più piccolo)
	 * 		è verificato!
	 * 	>
	 */

	extern __shared__ unsigned int  lm_sh[];

	// http://stackoverflow.com/questions/12505750/how-can-a-global-function-return-a-value-or-break-out-like-c-c-does
	__shared__ bool someoneFoundIt;

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;

	unsigned int otid	= bdx * r + c;

	unsigned int cc_0	= bdx*bdy*0;
	unsigned int nn_0	= bdx*bdy*1;
	unsigned int ww_0	= bdx*bdy*2;
	unsigned int ee_0	= bdx*bdy*3;
	unsigned int ss_0	= bdx*bdy*4;
	unsigned int ccR_0	= bdx*bdy*5;

	// CC tile
	unsigned int cc_tid	=	(r * gridDim.x * blockDim.x + c) 			+					// dentro la 1° tile		blockDim.x*(blockDim.y-1)+threadIdx.x
							(blockDim.x - 0) * (iTile % gridDim.x)		+					// itile in orizzontale		0
							(iTile / gridDim.x) * (blockDim.y-0) * gridDim.x * blockDim.x;	// itile in verticale		0

	// NN tile
	int nn_tile			= (iTile < gridDim.x)?-1:(iTile - gridDim.x);						// tile index of nn
	int nn_tid			=	(r * gridDim.x * blockDim.x + c) 			+					// dentro la 1° tile
							(blockDim.x - 0) * (nn_tile % gridDim.x)	+					// itile in orizzontale
							(nn_tile / gridDim.x) * (blockDim.y-0) * gridDim.x * blockDim.x;// itile in verticale   192

	// WW tile
	// see in MATLAB ==> reshape(mod(0:gridDim.x*gridDim.y-1,blockDim.x),blockDim.x,blockDim.y)'
	int ww_tile			= ((iTile % gridDim.x)==0)?-1:iTile-1;								// tile index of ww			9-1=8
	int ww_tid			=	(r * gridDim.x * blockDim.x + c) 			+					// dentro la 1° tile		r*10*5 +c ==> tile ad ovest della 9
							(blockDim.x - 0) * (ww_tile % gridDim.x)	+					// itile in orizzontale		5*mod(8,10)=40
							(ww_tile / gridDim.x) * (blockDim.y-0) * gridDim.x * blockDim.x;// itile in verticale		int(8/10)*5*10*5=0

	// SS tile
	int ss_tile			= (iTile >= gridDim.x*(gridDim.y-1))?-1:(iTile + gridDim.x);		// tile index of ss
	int ss_tid			=	(r * gridDim.x * blockDim.x + c) 			+					// dentro la 1° tile
							(blockDim.x - 0) * (ss_tile % gridDim.x)	+					// itile in orizzontale
							(ss_tile / gridDim.x) * (blockDim.y-0) * gridDim.x * blockDim.x;// itile in verticale

	// EE tile
	int ee_tile			= ((iTile % gridDim.x)==gridDim.x-1)?-1:iTile+1;					// tile index of ee
	int ee_tid			=	(r * gridDim.x * blockDim.x + c) 			+					// dentro la 1° tile
							(blockDim.x - 0) * (ee_tile % gridDim.x)	+					// itile in orizzontale
							(ee_tile / gridDim.x) * (blockDim.y-0) * gridDim.x * blockDim.x;// itile in verticale

	int ii=NULL;
	*gfound = false;
	if( iTile < gridDim.x*gridDim.y )
	{
		lm_sh[cc_0+otid] 					= lm[cc_tid]; __syncthreads();
		if( ww_tile>=0 ){ lm_sh[ww_0+otid] 	= lm[ww_tid]; __syncthreads(); }
		if( nn_tile>=0 ){ lm_sh[nn_0+otid] 	= lm[nn_tid]; __syncthreads(); }
		if( ee_tile>=0 ){ lm_sh[ee_0+otid] 	= lm[ee_tid]; __syncthreads(); }
		if( ss_tile>=0 ){ lm_sh[ss_0+otid] 	= lm[ss_tid]; __syncthreads(); }
		lm_sh[ccR_0+otid] 					= lm[cc_tid]; __syncthreads();

		someoneFoundIt = true;
		while(someoneFoundIt)
		{
			someoneFoundIt = false;

			// (1) objects_stitching_nn()
			if( nn_tile>=0 )
			{
				for(ii=0;ii<blockDim.x;ii++)
				{
					if( lm_sh[ccR_0+otid]==lm_sh[ccR_0+ii] )
					{
						if( lm_sh[nn_0+blockDim.x*(blockDim.y-1)+ii] < lm_sh[ccR_0+ii] )
						{
							lm_sh[cc_0+otid] = lm_sh[nn_0+blockDim.x*(blockDim.y-1)+ii];
							someoneFoundIt = true;
							__syncthreads();
						}
					}
				}
				//lm_sh[ccR_0+otid] = lm_sh[cc_0+otid];
				//__syncthreads();
			}

			// (2) objects_stitching_ww
			if( ww_tile>=0 )
			{
				for(ii=0;ii<blockDim.y;ii++)
				{
					if( lm_sh[ccR_0+otid]==lm_sh[ccR_0+blockDim.x*ii] )
					{
						if(lm_sh[ww_0+blockDim.x*(ii+1)-1] < lm_sh[ccR_0+blockDim.x*ii])
						{
							lm_sh[cc_0+otid] = lm_sh[ww_0+blockDim.x*(ii+1)-1];
							someoneFoundIt = true;
							__syncthreads();
						}
					}
				}
				//lm_sh[ccR_0+otid] = lm_sh[cc_0+otid];
				//__syncthreads();
			}

			// (3) objects_stitching_ee
			if( ee_tile>=0 )
			{
				for(ii=0;ii<blockDim.y;ii++)
				{
					if( lm_sh[ccR_0+otid]==lm_sh[ccR_0+blockDim.x*(ii+1)-1] )
					{
						if(lm_sh[ee_0+blockDim.x*ii] < lm_sh[ccR_0+blockDim.x*(ii+1)-1])
						{
							lm_sh[cc_0+otid] = lm_sh[ee_0+blockDim.x*ii];
							someoneFoundIt = true;
							__syncthreads();
						}
					}
				}
				//lm_sh[ccR_0+otid] = lm_sh[cc_0+otid];
				//__syncthreads();
			}

			// (4) objects_stitching_ss()
			if( ss_tile>=0 )
			{
				for(ii=0;ii<blockDim.x;ii++)
				{
					if( lm_sh[ccR_0+otid]==lm_sh[ccR_0+blockDim.x*(blockDim.y-1)+ii] )
					{
						if( lm_sh[ss_0+ii] < lm_sh[ccR_0+blockDim.x*(blockDim.y-1)+ii] )
						{
							lm_sh[cc_0+otid] = lm_sh[ss_0+ii];
							someoneFoundIt = true;
							__syncthreads();
						}
					}
				}
				//lm_sh[ccR_0+otid] = lm_sh[cc_0+otid];
				//__syncthreads();
			}
			//__syncthreads();
			if(someoneFoundIt) *gfound = true;
			lm_sh[ccR_0+otid] = lm_sh[cc_0+otid];
			__syncthreads();
		}//while :: single cc
	}//blocks	 :: all cc's

	// I write the borders of all tiles synchronous without knowing
	// which tile has the lowest ID in any border and any pixel because
	// I preserve whole tile info with all duplicated borders.
	lm[cc_tid] = lm_sh[cc_0+otid];
	__syncthreads();

}//kernel

__global__ void linearize_tiles( unsigned char *urban, unsigned int NC )
{
	/*
	 * 	NOTE: I am not sure that for larger image sizes this kernel works fine!!
	 */
	extern __shared__ unsigned char  urban_sh[];

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;

	//unsigned int NC		= (bdx-1) * gdx;
	unsigned int otid		= bdx * r + c;
	unsigned int itid		= (r * NC + c) 					+		// (3) within-tile	offset
							  (bdx - 1) * (iTile % gdx)		+		// (2) horizontal 	offset
							  (iTile / gdx) * (bdy-1) * NC;			// (1) vertical 	offset
	unsigned int ttid 		= iTile*bdx*bdy+otid;					// linearized tiles + 1-pixel-width extra border along the tile perimeter.
	unsigned int stid		= (r * gdx * bdx + c) 			+		// (3) within-tile	offset
							  (bdx - 0) * (iTile % gdx)		+		// (2) horizontal 	offset
							  (iTile / gdx) * (bdy-0) * gdx * bdx;	// (1) vertical 	offset

	if (iTile<gdx*gdy)
	{
		// why not urban[ttid] = urban[itid] directly??
		urban_sh[otid] 	= urban[itid];		__syncthreads();
		urban[ttid]		= urban_sh[otid];	__syncthreads();
	}
}


__global__ void intra_tile_labeling(const unsigned char *urban,unsigned int NC,unsigned int *lab_mat)
{
	// See this link when using more then one extern __shared__ array:
	// 		http://stackoverflow.com/questions/9187899/cuda-shared-memory-array-variable
	extern __shared__ unsigned int  lab_mat_sh[];

	__shared__ bool found;

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;

	unsigned int otid		= bdx * r + c;
	unsigned int itid		= (r * NC + c) 					+		// dentro la 1° tile		0
							  (bdx - 1) * (iTile % gdx)		+		// itile in orizzontale		0
							  (iTile / gdx) * (bdy-1) * NC;			// itile in verticale		0
	unsigned int ttid 		= iTile*bdx*bdy+otid;
	unsigned int stid		= (r * gdx * bdx + c) 			+		// dentro la 1° tile		0
			  	  	  	  	  (bdx - 0) * (iTile % gdx)		+		// itile in orizzontale		0
			  	  	  	  	  (iTile / gdx) * (bdy-0) * gdx * bdx;	// itile in verticale		42

	if (iTile<gdx*gdy)
	{
		lab_mat_sh[otid] 	= 0;
		// if (r,c) is object pixel
		//if  (urban[ttid]==Vo)  lab_mat_sh[otid] = ttid; // use ttid with 	"linearize_tiles"
		if  (urban[itid]==Vo)  lab_mat_sh[otid] = ttid; // use itid without "linearize_tiles"
		__syncthreads();

		found = true;
		while(found)
		{
			/* 		________________
			 * 		|	 |    |    |
			 *		| nw | nn | ne |
			 *		|____|____|____|
			 * 		|	 |    |    |
			 * 		| ww | cc | ee |	pixel position
			 *		|____|____|____|
			 * 		|	 |    |    |
			 * 		| sw | ss | se |
			 * 		|____|____|____|
			 */
			found = false;

			// NW:
			if(	c>0 && r>0 && nw_pol(c,r)!=0 && nw_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = nw_pol(c,r); found = true; }
			// NN:
			if( r>0 && nn_pol(c,r)!=0 && nn_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = nn_pol(c,r); found = true; }
			// NE:
			if( c<bdx-1 && r>0 && ne_pol(c,r)!=0 && ne_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = ne_pol(c,r); found = true; }
			// WW:
			if( c>0 && ww_pol(c,r)!=0 && ww_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = ww_pol(c,r); found = true; }
			// EE:
			if( c<bdx-1 && ee_pol(c,r)!=0 && ee_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = ee_pol(c,r); found = true; }
			// SW:
			if( c>0 && r<bdy-1 && sw_pol(c,r)!=0 && sw_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = sw_pol(c,r); found = true; }
			// SS:
			if( r<bdy-1 && ss_pol(c,r)!=0 && ss_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = ss_pol(c,r); found = true; }
			// SE:
			if( c<bdx-1 && r<bdy-1 && se_pol(c,r)!=0 && se_pol(c,r)<cc_pol(c,r))
				{ cc_pol(c,r) = se_pol(c,r); found = true; }

			__syncthreads();
		}

		/*
		 * 	To linearize I write using ttid.
		 * 	To leave same matrix configuration as input urban use stid instead!!
		 */
		lab_mat[ttid] = lab_mat_sh[otid];
		//__syncthreads();
	}
}

__global__ void intra_tile_labeling_opt(const unsigned char *urban,unsigned int NC,unsigned int *lab_mat)
{
	extern __shared__ unsigned int  lab_mat_sh[];
	__shared__ bool found[1];

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;

	unsigned char urban_loc;
	unsigned char neigh_loc[8];
	unsigned int fill_val 	= 0xFFFFFFFFFFFFFFFF;
	unsigned int newLabel;
	unsigned int oldLabel;

	unsigned int otid		= bdx * r + c;
	unsigned int itid		= (r * NC + c) 					+		// dentro la 1° tile		0
							  (bdx - 1) * (iTile % gdx)		+		// itile in orizzontale		0
							  (iTile / gdx) * (bdy-1) * NC;			// itile in verticale		0
	unsigned int ttid 		= iTile*bdx*bdy+otid;
/*	unsigned int stid		= (r * gdx * bdx + c)			+		// dentro la 1° tile		0
			  	  	  	  	  (bdx - 0) * (iTile % gdx)		+		// itile in orizzontale		0
			  	  	  	  	  (iTile / gdx) * (bdy-0) * gdx * bdx;	// itile in verticale		42
*/
	unsigned int ex_tid		= c+1 + (r+1)*(bdx+2);

	if (iTile<gdx*gdy)
	{
		// initialize with maximum value:
		lab_mat_sh[ex_tid] 		= fill_val;

/*
		// initialize with maximum value:
		lab_mat_sh[ex_tid] 						= fill_val;
		// **load all zeros in boundaries:
		if(c==0 && r==0){
			lab_mat_sh[0] 						= fill_val;
			lab_mat_sh[bdx+1] 					= fill_val;
			lab_mat_sh[(bdx+2)*(bdy+1)] 		= fill_val;
			lab_mat_sh[(bdx+2)*(bdy+2) -1] 		= fill_val;
		}
		if(c<bdx)
		{
			lab_mat_sh[c+1] 					= fill_val;
			lab_mat_sh[(bdx+2)*(bdy+1)+1 +c] 	= fill_val;
		}
		if(r<bdy)
		{
			lab_mat_sh[(r+1)*(bdx+2)]			= fill_val;
			lab_mat_sh[(r+1)*(bdx+2)+bdx+1] 	= fill_val;
		}
		//****
*/
/*
		// fill with fill_value
		lab_mat_sh[ ex_tid - (bdx+2) 	-1 ] = fill_val;
		lab_mat_sh[ ex_tid - (bdx+2) 	+0 ] = fill_val;
		lab_mat_sh[ ex_tid - (bdx+2) 	+1 ] = fill_val;
		lab_mat_sh[ ex_tid 			-1 ]	 = fill_val;
		lab_mat_sh[ ex_tid			+1 ]	 = fill_val;
		lab_mat_sh[ ex_tid + (bdx+2) 	-1 ] = fill_val;
		lab_mat_sh[ ex_tid + (bdx+2) 	+0 ] = fill_val;
		lab_mat_sh[ ex_tid + (bdx+2) 	+1 ] = fill_val;
		__syncthreads();
*/

		// use per-thread memory facility:
		urban_loc 	 			= urban[itid];//(unsigned char)lab_mat_sh;
		// load binary objects:
		if( urban_loc==Vo ) lab_mat_sh[ex_tid] = urban_loc; /*if( urban_loc!=Vb )*/
		__syncthreads();

		neigh_loc[0] 			= lab_mat_sh[ ex_tid - (bdx+2) 	-1 ];
		neigh_loc[1] 			= lab_mat_sh[ ex_tid - (bdx+2) 	+0 ];
		neigh_loc[2] 			= lab_mat_sh[ ex_tid - (bdx+2) 	+1 ];
		neigh_loc[3] 			= lab_mat_sh[ ex_tid 			-1 ];
		neigh_loc[4] 			= lab_mat_sh[ ex_tid			+1 ];
		neigh_loc[5] 			= lab_mat_sh[ ex_tid + (bdx+2) 	-1 ];
		neigh_loc[6] 			= lab_mat_sh[ ex_tid + (bdx+2) 	+0 ];
		neigh_loc[7] 			= lab_mat_sh[ ex_tid + (bdx+2) 	+1 ];

		// load global unique index:
		newLabel 				= ttid;
		if( urban_loc==Vo ) lab_mat_sh[ex_tid] = newLabel; /*if( urban_loc!=Vb )*/
		while( 1 ){
			found[0] 			= false;
			oldLabel 			= newLabel;
			__syncthreads();

			if(urban_loc != Vb){
/*				if(neigh_loc[0]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid - (bdx+2)	-1 ]);
				if(neigh_loc[1]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid - (bdx+2)	+0 ]);
				if(neigh_loc[2]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid - (bdx+2)	+1 ]);
				if(neigh_loc[3]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid			-1 ]);
				if(neigh_loc[4]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid			+1 ]);
				if(neigh_loc[5]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid + (bdx+2)	-1 ]);
				if(neigh_loc[6]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid + (bdx+2)	+0 ]);
				if(neigh_loc[7]==Vo) newLabel = min(newLabel, lab_mat_sh[ ex_tid + (bdx+2)	+1 ]);
*/				newLabel = min(newLabel, lab_mat_sh[ ex_tid - (bdx+2) -1 ]);
				newLabel = min(newLabel, lab_mat_sh[ ex_tid - (bdx+2) +0 ]);
				newLabel = min(newLabel, lab_mat_sh[ ex_tid - (bdx+2) +1 ]);
				newLabel = min(newLabel, lab_mat_sh[ ex_tid 		  -1 ]);
				newLabel = min(newLabel, lab_mat_sh[ ex_tid			  +1 ]);
				newLabel = min(newLabel, lab_mat_sh[ ex_tid + (bdx+2) -1 ]);
				newLabel = min(newLabel, lab_mat_sh[ ex_tid + (bdx+2) +0 ]);
				newLabel = min(newLabel, lab_mat_sh[ ex_tid + (bdx+2) +1 ]);
			}
			__syncthreads();

			if(oldLabel > newLabel) {
				atomicMin(&lab_mat_sh[ex_tid], newLabel); // if it is slow ==> write directly!
				//lab_mat_sh[ex_tid] = newLabel;
				//set the flag to 1 -> it is necessary to perform another iteration of the CCL solver
				found[0] 		= true;
			}
			__syncthreads();
			//if no equivalence was updated, the local solution is complete
			if(found[0] == false) break;
		}

		/*  To linearize write using ttid.
		 * 	To leave same matrix configuration as input urban use stid instead!!
		 */
		if( urban_loc==Vo ) lab_mat[ttid] = lab_mat_sh[ex_tid];
		__syncthreads();
	}
}
__global__ void intra_tile_labeling_opt2(const unsigned char *urban,unsigned int NC,unsigned int *lab_mat)
{
	extern __shared__ unsigned int  lab_mat_sh[];
	__shared__ bool found;

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;
	unsigned int nTiles		= gdx * gdy;

	unsigned char urb_cc_loc;
	unsigned int lab_neigh_loc[8];
	unsigned int fill_val 	= 0xFFFFFFFFFFFFFFFF;
	unsigned int newLabel;
	unsigned int oldLabel;

	unsigned int otid		= bdx * r + c;
	unsigned int itid		= (r * NC + c) 					+		// dentro la 1° tile		0
							  (bdx - 1) * (iTile % gdx)		+		// itile in orizzontale		0
							  (iTile / gdx) * (bdy-1) * NC;			// itile in verticale		0
	unsigned int ttid 		= iTile*bdx*bdy+otid;
/*	unsigned int stid		= (r * gdx * bdx + c)			+		// dentro la 1° tile		0
			  	  	  	  	  (bdx - 0) * (iTile % gdx)		+		// itile in orizzontale		0
			  	  	  	  	  (iTile / gdx) * (bdy-0) * gdx * bdx;	// itile in verticale		42
*/
	unsigned int ex_tid		= c+1 + (r+1)*(bdx+2);

	unsigned int ii			= 0;

	if( iTile<nTiles )
	{
		// initialize with maximum value:
		lab_mat_sh[ex_tid] 						= fill_val;
		/* **write fill_val in boundaries** */
		if( c==0 && r==0 ){		//..:: 4x corners ::..
			lab_mat_sh[0] 						= fill_val;
			lab_mat_sh[bdx+1] 					= fill_val;
			lab_mat_sh[(bdx+2)*(bdy+1)] 		= fill_val;
			lab_mat_sh[(bdx+2)*(bdy+2) -1] 		= fill_val;
		}
		if( c<bdx ) {			//..:: nn+ss ::..
			lab_mat_sh[c+1] 					= fill_val;
			lab_mat_sh[(bdx+2)*(bdy+1)+1 +c] 	= fill_val;
		}
		if( r<bdy ){			//..:: ww+ee ::..
			lab_mat_sh[(r+1)*(bdx+2)]			= fill_val;
			lab_mat_sh[(r+1)*(bdx+2)+bdx+1] 	= fill_val;
		}

		// use per-thread memory facility:
		urb_cc_loc 	 							= urban[itid];//(unsigned char)lab_mat_sh;
		__syncthreads();

		// load global unique index:
		if( urb_cc_loc==Vo ) lab_mat_sh[ex_tid] = ttid; /*if( urb_cc_loc!=Vb )*/
		__syncthreads();

		// if no equivalence was updated, the local solution is complete
		newLabel 					= lab_mat_sh[ex_tid];
		found						= true;
		while( found==true ){
			found 					= false;
			oldLabel 				= newLabel;
			// for each thread load the 8-adjacent pixels
			lab_neigh_loc[0] 		= lab_mat_sh[ ex_tid - (bdx+2) 	-1 ];
			lab_neigh_loc[1] 		= lab_mat_sh[ ex_tid - (bdx+2) 	+0 ];
			lab_neigh_loc[2] 		= lab_mat_sh[ ex_tid - (bdx+2) 	+1 ];
			lab_neigh_loc[3] 		= lab_mat_sh[ ex_tid 			-1 ];
			lab_neigh_loc[4] 		= lab_mat_sh[ ex_tid			+1 ];
			lab_neigh_loc[5] 		= lab_mat_sh[ ex_tid + (bdx+2) 	-1 ];
			lab_neigh_loc[6] 		= lab_mat_sh[ ex_tid + (bdx+2) 	+0 ];
			lab_neigh_loc[7] 		= lab_mat_sh[ ex_tid + (bdx+2) 	+1 ];

			for(ii=0;ii<8;ii++){
				newLabel 			= fminf( newLabel, lab_neigh_loc[ii] );
			}
			//atomicMin(&lab_mat_sh[ex_tid], newLabel);
			if( urb_cc_loc==Vo ) lab_mat_sh[ex_tid] = newLabel;

			//set the flag to 1 -> it is necessary to perform another iteration of the CCL solver
			if(oldLabel > newLabel){ found = true; }
			__syncthreads();
		}

		/*  To linearize write using ttid.
		 * 	To leave same matrix configuration as input urban use stid instead!!
		 */
		if( urb_cc_loc==Vo ) lab_mat[ttid] = lab_mat_sh[ex_tid];
		__syncthreads();
	}
}

template <unsigned int NTHREADSX>
__global__ void stitching_tiles(	unsigned int *lab_mat,
									const unsigned int tiledimX,
									const unsigned int tiledimY		)
{
	/*
	 * 	NOTE:
	 * 		> xx_yy is the tile xx and border yy (e.g. nn_ss is tile at north and border at south).
	 */

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;

	// TILES:
	int nTiles			= gdx * gdy;
	int nn_tile			= (iTile < gdx)?-1:(iTile - gdx);		// nn tile of cc tile
	int ww_tile			= ((iTile % gdx)==0)?-1:iTile-1;		// ww tile of cc tile

	// SIDES:
	int c_nn_tid		=	c 								+	// (2) within-tile
							tiledimX*tiledimY * iTile;			// (1) horizontal offset

	int nn_tid			= 	c +	tiledimX*(tiledimY-1) 		+	// (2) within-tile
							tiledimX*tiledimY * nn_tile;		// (1) horizontal offset

	int c_ww_tid		=	c*tiledimX 						+	// (2) within-tile
							tiledimX*tiledimY * iTile;			// (1) horizontal offset

	int ww_tid			= 	(c+1)*tiledimX-1 				+	// (2) within-tile
							tiledimX*tiledimY * ww_tile;		// (1) horizontal offset

	// SHARED: "tile_border" ==> cc_nn is border North of Center tile
	__shared__ unsigned int cc_nn[NTHREADSX];
	__shared__ unsigned int nn_ss[NTHREADSX];
	__shared__ unsigned int cc_ww[NTHREADSX];
	__shared__ unsigned int ww_ee[NTHREADSX];
	__shared__ unsigned int __old[NTHREADSX];
	__shared__ unsigned int _min_[NTHREADSX];
	__shared__ unsigned int _max_[NTHREADSX];

	if( iTile < nTiles )
	{
		// ...::NORTH::...
		if( nn_tile>=0 ){
			//recursion ( lab_mat, c_nn_tid, nn_tid );
			/*
			 * 		(1) **list** { cc_nn(i), nn_ss(i) }
			 */
			cc_nn[ c ]		= lab_mat[ c_nn_tid ];
			nn_ss[ c ] 		= lab_mat[ nn_tid ];
			__syncthreads();
			/*
			 * 		(2) **recursion applying split-rules**
			 */
			__old[ c ] = atomicMin( &lab_mat[ cc_nn[c] ], nn_ss[ c ] ); // write the current min val where the index cc_nn[c] is in lab_mat.
			while( __old[ c ] != nn_ss[c] )
			{
				_min_[ c ] 	= ( (nn_ss[c]) < (__old[c]) )? nn_ss[c] : __old[c];
				_max_[ c ] 	= ( (nn_ss[c]) > (__old[c]) )? nn_ss[c] : __old[c];
				__old[ c ] 	= atomicMin( &lab_mat[ _max_[c] ], _min_[ c ] );
				nn_ss[ c ] 	= _min_[ c ];
			}
			__syncthreads();
		}

		// ...::WEST::...
		if( ww_tile>=0 ){
			//recursion ( lab_mat, c_ww_tid, ww_tid );
			/*
			 * 		(1) **list** { cc_nn(i), nn_ss(i) }
			 */
			cc_ww[ c ]		= lab_mat[ c_ww_tid ];
			ww_ee[ c ] 		= lab_mat[ ww_tid ];
			__syncthreads();

			/*
			 * 		(2) **recursion applying split-rules**
			 */
			__old[ c ] = atomicMin( &lab_mat[ cc_ww[c] ], ww_ee[ c ] );
			//__syncthreads();
			while( __old[ c ] != ww_ee[c] )
			{
				_min_[ c ] 	= ( (ww_ee[c]) < (__old[c]) )? (ww_ee[c]) : (__old[c]);
				_max_[ c ] 	= ( (ww_ee[c]) > (__old[c]) )? (ww_ee[c]) : (__old[c]);
				__old[ c ] 	= atomicMin( &lab_mat[ _max_[c] ], _min_[ c ] );
				ww_ee[ c ] 	= _min_[ c ];//lab_mat[ _max_[c] ];
			}
			__syncthreads();
		}
	}
}

template <unsigned int NTHREADSX>
__global__ void root_equivalence(	unsigned int *lab_mat,
									const unsigned int tiledimX,
									const unsigned int tiledimY		)
{

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;

	// TILES:
	int nTiles			= gdx * gdy;
	int nn_tile			= (iTile < gdx)?-1:(iTile - gdx);			// nn tile of cc tile
	int ww_tile			= ((iTile % gdx)==0)?-1:iTile-1;			// ww tile of cc tile
	int ss_tile			= (iTile >= gdx*(gdy-1))?-1:(iTile + gdx);	// tile index of ss
	int ee_tile			= ((iTile % gdx)==gdx-1)?-1:iTile+1;		// tile index of ee

	// SIDES:
	int c_nn_tid		=	c 								+		// (2) within-tile
							tiledimX*tiledimY * iTile;				// (1) horizontal offset
	int nn_tid			= 	c + tiledimX*(tiledimY-1)		+		// (2) within-tile
							tiledimX*tiledimY * nn_tile;			// (1) horizontal offset
	int c_ww_tid		=	c*tiledimX 						+		// (2) within-tile
							tiledimX*tiledimY * iTile;				// (1) horizontal offset
	int ww_tid			= 	(c+1)*tiledimX-1 				+		// (2) within-tile
							tiledimX*tiledimY * ww_tile;			// (1) horizontal offset

	int c_ss_tid		=	c + tiledimX*(tiledimY-1)		+		// (2) within-tile
							tiledimX*tiledimY * iTile;				// (1) horizontal offset
	int ss_tid			= 	c								+		// (2) within-tile
							tiledimX*tiledimY * ss_tile;			// (1) horizontal offset
	int c_ee_tid		=	(c+1)*tiledimX-1				+		// (2) within-tile
							tiledimX*tiledimY * iTile;				// (1) horizontal offset
	int ee_tid			= 	c*tiledimX 						+		// (2) within-tile
							tiledimX*tiledimY * ee_tile;			// (1) horizontal offset

	// SHARED:
	__shared__ unsigned int cc_nn[NTHREADSX];
	__shared__ unsigned int nn_ss[NTHREADSX];
	__shared__ unsigned int cc_ww[NTHREADSX];
	__shared__ unsigned int ww_ee[NTHREADSX];
	__shared__ unsigned int cc_ss[NTHREADSX];
	__shared__ unsigned int ss_nn[NTHREADSX];
	__shared__ unsigned int cc_ee[NTHREADSX];
	__shared__ unsigned int ee_ww[NTHREADSX];

	if( iTile < nTiles )
	{
		// ...::NORTH::...
		if( nn_tile>=0 ){
			/*
			 * 		(1) **list** { cc_nn(i), nn_ss(i) }
			 */
			cc_nn[ c ]		= lab_mat[ c_nn_tid ];
			nn_ss[ c ] 		= lab_mat[ nn_tid ]; // --> DELETE, because nn_cc = cc_nn after!!
			__syncthreads();

			/*
			 * 		(2) **recursion finding root equivalence** nn_ss(i) = lab_mat[ nn_ss(i) ]

						lab_mat[ nn_ss(i) ]  ---> ID(t)   ------|
																| 	with recursion on t
						lab_mat[ ID(t) ]     ---> ID(t+1) <-----|
			 */
			nn_ss[ c ]		= cc_nn[c];
			while( lab_mat[ nn_ss[c] ] != nn_ss[c] )
			{
				{
					nn_ss[ c ] 	= lab_mat[ nn_ss[c] ];
					__threadfence_system();//__syncthreads();
				}
			}
			atomicMin( &lab_mat[ cc_nn[ c ] ], nn_ss[c] );
			atomicMin( &lab_mat[ c_nn_tid ],   nn_ss[c] );
			//lab_mat[ c_nn_tid ] = lab_mat[ nn_ss[c] ];
			__syncthreads();
		}

		// ...::WEST::...
		if( ww_tile>=0 ){
			/*
			 * 		(1) **list** { cc_nn(i), nn_ss(i) }
			 */
			cc_ww[ c ]		= lab_mat[ c_ww_tid ];
			ww_ee[ c ] 		= lab_mat[ ww_tid ];
			__syncthreads();

			/*
			 * 		(2) **recursion finding root equivalence** nn_ss(i) = lab_mat[ nn_ss(i) ]

						lab_mat[ nn_ss(i) ]  ---> ID(t)   ------|
																| 	with recursion on t
						lab_mat[ ID(t) ]     ---> ID(t+1) <-----|
			 */
			ww_ee[ c ]		= cc_ww[c];
			while( lab_mat[ ww_ee[c] ] != ww_ee[c] )
			{
				ww_ee[ c ] 	= lab_mat[ ww_ee[c] ];
				__threadfence_system();//__syncthreads();
			}
			atomicMin( &lab_mat[ cc_ww[ c ] ], ww_ee[c] );
			atomicMin( &lab_mat[ c_ww_tid ],   ww_ee[c] );
			//lab_mat[ c_ww_tid ] = lab_mat[ ww_ee[c] ];
			__syncthreads();//__threadfence_system();
		}

		// ...::SOUTH::...
		if( ss_tile>=0 ){
			/*
			 * 		(1) **list** { cc_nn(i), nn_ss(i) }
			 */
			cc_ss[ c ]		= lab_mat[ c_ss_tid ];
			ss_nn[ c ] 		= lab_mat[ ss_tid ];
			__syncthreads();

			/*
			 * 		(2) **recursion finding root equivalence** nn_ss(i) = lab_mat[ nn_ss(i) ]

						lab_mat[ nn_ss(i) ]  ---> ID(t)   ------|
																| 	with recursion on t
						lab_mat[ ID(t) ]     ---> ID(t+1) <-----|
			 */
			ss_nn[ c ]		= cc_ss[c];
			while( lab_mat[ ss_nn[c] ] != ss_nn[c] )
			{
				ss_nn[ c ] 	= lab_mat[ ss_nn[c] ];
				__threadfence_system();//__syncthreads();
			}
			atomicMin( &lab_mat[ cc_ss[ c ] ], ss_nn[c] );
			atomicMin( &lab_mat[ c_ss_tid ],   ss_nn[c] );
			__syncthreads();
		}

		// ...::EAST::...
		if( ee_tile>=0 ){
			/*
			 * 		(1) **list** { cc_nn(i), nn_ss(i) }
			 */
			cc_ee[ c ]		= lab_mat[ c_ee_tid ];
			ee_ww[ c ] 		= lab_mat[ ee_tid ];
			__syncthreads();

			/*
			 * 		(2) **recursion finding root equivalence** nn_ss(i) = lab_mat[ nn_ss(i) ]

						lab_mat[ nn_ss(i) ]  ---> ID(t)   ------|
																| 	with recursion on t
						lab_mat[ ID(t) ]     ---> ID(t+1) <-----|
			 */
			ee_ww[ c ]		= cc_ee[c];
			while( lab_mat[ ee_ww[c] ] != ee_ww[c] )
			{
				ee_ww[ c ] 	= lab_mat[ ee_ww[c] ];
				__threadfence_system();//__syncthreads();
			}
			atomicMin( &lab_mat[ cc_ee[ c ] ], ee_ww[c] );
			atomicMin( &lab_mat[ c_ee_tid ],   ee_ww[c] );
			__syncthreads();//__threadfence_system();
		}
	}
}

__global__ void intra_tile_re_label(unsigned int NC,unsigned int *lab_mat)
{
	// See this link when using more then one extern __shared__ array:
	// 		http://stackoverflow.com/questions/9187899/cuda-shared-memory-array-variable
	//extern __shared__ unsigned char urban_sh[];
//	extern __shared__ unsigned int  lab_mat_sh[];

//	__shared__ bool found;

	unsigned int r 			= threadIdx.y;
	unsigned int c 			= threadIdx.x;
	unsigned int bdx		= blockDim.x;
	unsigned int bdy		= blockDim.y;
	unsigned int bix		= blockIdx.x;
	unsigned int biy		= blockIdx.y;
	unsigned int gdx		= gridDim.x;
	unsigned int gdy		= gridDim.y;
	unsigned int iTile		= gdx * biy + bix;

	unsigned int otid		= bdx * r + c;
	unsigned int itid		= (r * NC + c) 						+	// dentro la 1° tile		0
							  (bdx - 1) * (iTile % gdx)			+	// itile in orizzontale		0
							  (iTile / gdx) * (bdy-1) * NC;			// itile in verticale		0
	unsigned int ttid 		= iTile*bdx*bdy+otid;
	unsigned int stid		= (r * gdx * bdx + c) 		+			// dentro la 1° tile		0
			  	  	  	  	  (bdx - 0) * (iTile % gdx)			+	// itile in orizzontale		0
			  	  	  	  	  (iTile / gdx) * (bdy-0) * gdx * bdx;	// itile in verticale		42

	if (iTile<gdx*gdy)
	{
		// try to write a sequence of IDs starting from 1 to N found labels!!
		// ...some code...

		if  (lab_mat[ttid]!=Vb)  lab_mat[ttid]=lab_mat[lab_mat[ttid]];
		//if  (urban[itid]==Vo)  urban[itid]=lab_mat[lab_mat[ttid]];
	}
}

int main(int argc, char **argv)
{
	// INPUTS
	unsigned int tiledimX 	= atoi( argv[1] );	// tile dim in X
	unsigned int tiledimY 	= atoi( argv[2] );	// tile dim in Y
	unsigned int NC1 		= atoi( argv[3] );	// ncols
	unsigned int NR1 		= atoi( argv[4] );	// nrows
	unsigned int printme	= atoi( argv[5] );	// nrows

	const unsigned int NTHREADSX = 32;			// how to let it be variable.

	if( NTHREADSX!=tiledimX ){
		fprintf(stderr, "Error: NTHREADSX(=%d) <> tiledimX(=%d)!\n", NTHREADSX,tiledimX);
		printf("\t[modify it according to tiledimX(=%d) (which should be equal to tiledimY(=%d)!!]\n\n",tiledimX,tiledimY);
		exit(EXIT_FAILURE);
	}

	// count the number of kernels that must print their LAB-MAT:
	unsigned int count_print=0;

	// MANIPULATION:
	// X dir
	unsigned int ntilesX 	= ceil( (double)(NC1+2-1) / (double)(tiledimX-1)  );
	unsigned int NC 		= ntilesX*(tiledimX-1) +1;// number of columns 	of URBAN with 1-pixel-widht zero perimeter
	// Y dir
	unsigned int ntilesY	= ceil( (double)(NR1+2-1) / (double)(tiledimY-1)  );
	unsigned int NR 		= ntilesY*(tiledimY-1) +1;// number of rows		of URBAN with 1-pixel-widht zero perimeter
/*	printf("nTiles.X: %d\nnTiles.Y: %d\n",ntilesX,ntilesY);
	printf("NR:       %d\nNC:       %d\n",NR,NC);
	printf("tileDim.X: %d\ntileDim.Y: %d\n\n",tiledimX,tiledimY);
*/
	// DECLARATIONS:
	//	Error code to check return values for CUDA calls
	hipError_t cudaLastErr = hipSuccess;

	// size of arrays
	size_t sizeChar  		= NC*NR * sizeof(unsigned char);
	size_t sizeUintL 		= ntilesX*ntilesY*tiledimX*tiledimY * sizeof(unsigned int);
	// clocks:
	clock_t start_t, end_t;

	/*
		cudaStream_t stream[2];
		cudaStreamCreate(&stream[0]);
		cudaStreamCreate(&stream[1]);
	*/



	/* ....::: ALLOCATION :::.... */

	// -1- urban_cpu
	unsigned char *urban_cpu;
	hipHostMalloc(&urban_cpu,sizeChar, hipHostMallocDefault);
	read_urbmat(urban_cpu, NR, NC, ALL_txt);
	// -2- urban_gpu -- stream[0]
	unsigned char *urban_gpu;
	cudaLastErr = hipMalloc( (void **)&urban_gpu, sizeChar );
	if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to allocate device array urban_gpu (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	//cudaLastErr = cudaMemsetAsync( urban_gpu,0, sizeChar, stream[0] );
	cudaLastErr = hipMemset( urban_gpu,0, sizeChar );
	if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to set ZEROS in urban_gpu array on device (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	//cudaLastErr = cudaMemcpyAsync( urban_gpu,urban_cpu,	sizeChar,cudaMemcpyHostToDevice, stream[0] );
	cudaLastErr = hipMemcpy( urban_gpu,urban_cpu,	sizeChar,hipMemcpyHostToDevice );
	if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to copy array urban_cpu from host to device urban_gpu (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	// -3- lab_mat_cpu
	unsigned int  *lab_mat_cpu;
	hipHostMalloc(&lab_mat_cpu,sizeUintL, hipHostMallocDefault);
	// -4- lab_mat_gpu  -- stream[1]
	unsigned int  *lab_mat_gpu;
	cudaLastErr = hipMalloc( (void **)&lab_mat_gpu, sizeUintL );
	if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to allocate device array lab_mat_gpu (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	//cudaLastErr = cudaMemsetAsync( lab_mat_gpu,0, sizeUintL, stream[0] );
	cudaLastErr = hipMemset( lab_mat_gpu,0, sizeUintL );
	if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to set ZEROS in lab_mat_gpu array on device (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }

	/* ....::: ALLOCATION :::.... */



	start_t = clock();

/*
 *		KERNELS INVOCATION
 *
 *			*************************
 *			-1- linearize_tiles			|\
 *			-2- intra_tile_labeling		| --> 1st Stage
 *
 *			-3- stitching_tiles			|\
 *			-4- root_equivalence		| --> 2nd Stage
 *
 *			-5- intra_tile_re_label		| --> 3rd Stage
 *			*************************
 */

	/* ....::: [1/3 stage] INTRA-TILE :::.... */

	dim3 	block(tiledimX,tiledimY,1);
	dim3 	grid(ntilesX,ntilesY,1);
	int 	sh_mem	= (tiledimX*tiledimY)*(sizeof(unsigned int)); // +sizeof(unsigned char)
	int 	sh_mem_2= ((tiledimX+2)*(tiledimY+2))*(sizeof(unsigned int)); // +sizeof(unsigned char)

/*	linearize_tiles<<<grid,block,sh_mem>>>(urban_gpu,NC);
	cudaLastErr		= cudaGetLastError();
	if (cudaLastErr != cudaSuccess){ printf ("ERROR {linearize_tiles} -- %s\n", cudaGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
*/	/* INTERMEDIATE CHECK [activate/deactivate]*/
/*	printf("  -0- %30s\n","print original");
	sprintf(buffer,"/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/-0-urban_cpu-read_from_HDD.txt");
	write_urbmat_matlab(urban_cpu, tiledimY, tiledimX, ntilesX, ntilesY, buffer);
	printf("  -1- %30s\n","linearize_tiles");
	cudaLastErr 	= cudaMemcpy(urban_cpu,urban_gpu,	sizeChar,cudaMemcpyDeviceToHost);
	if (cudaLastErr != cudaSuccess){ fprintf(stderr, "Failed to allocate copy array urban_gpu from device to host (error code %s)!\n", cudaGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	sprintf(buffer,"/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/-1-urban_cpu-linearize_tiles.txt");
	write_urbmat_tiled(urban_cpu, tiledimY, tiledimX, ntilesX, ntilesY, buffer);
*/
	intra_tile_labeling<<<grid,block,sh_mem>>>(urban_gpu,NC,lab_mat_gpu);
//	intra_tile_labeling_opt<<<grid,block,sh_mem_2>>>(urban_gpu,NC,lab_mat_gpu);
//	intra_tile_labeling_opt2<<<grid,block,sh_mem_2>>>(urban_gpu,NC,lab_mat_gpu);
	cudaLastErr 	= hipGetLastError();
	if (cudaLastErr != hipSuccess){ printf ("ERROR {intra_tile_labeling} -- %s\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	/* INTERMEDIATE CHECK [activate/deactivate]*/
	if (printme){
		count_print++;
		printf("  -%d- %30s\n",count_print,"intra_tile_labeling");
		cudaLastErr 	= hipMemcpy(lab_mat_cpu,lab_mat_gpu,	sizeUintL,hipMemcpyDeviceToHost);
		if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to allocate copy array lab_mat_gpu from device to host (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
		sprintf(buffer,"/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/-2-lab_mat_cpu-intra_tile_labeling.txt");
		write_labmat_tiled(lab_mat_cpu, tiledimY, tiledimX, ntilesX, ntilesY, buffer);
	}
	/* ....::: [1/3 stage] :::.... */




	/* ....::: [2/3 stage] STITCHING :::.... */

	dim3 	block_2(tiledimX,1,1);
	dim3 	grid_2(ntilesX,ntilesY,1);

	stitching_tiles<NTHREADSX><<<grid_2,block_2>>>(lab_mat_gpu,tiledimX,tiledimY);
	cudaLastErr 	= hipGetLastError();
	if (cudaLastErr != hipSuccess){ printf ("ERROR {stitching_tiles} -- %s\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	/* INTERMEDIATE CHECK [activate/deactivate]*/
	if (printme){
		count_print++;
		printf("  -%d- %30s\n",count_print,"stitching_tiles");
		cudaLastErr 	= hipMemcpy(lab_mat_cpu,lab_mat_gpu,	sizeUintL,hipMemcpyDeviceToHost);
		if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to allocate copy array lab_mat_gpu from device to host (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
		sprintf(buffer,"/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/-3-lab_mat_cpu-stitching_tiles.txt");
		write_labmat_tiled(lab_mat_cpu, tiledimY, tiledimX, ntilesX, ntilesY, buffer);
	}

	root_equivalence<NTHREADSX><<<grid_2,block_2>>>(lab_mat_gpu,tiledimX,tiledimY);
	cudaLastErr 	= hipGetLastError();
	if (cudaLastErr != hipSuccess){ printf ("ERROR {root_equivalence} -- %s\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	/* INTERMEDIATE CHECK [activate/deactivate]*/
	if (printme){
		count_print++;
		printf("  -%d- %30s\n",count_print,"stitching_tiles");
		cudaLastErr 	= hipMemcpy(lab_mat_cpu,lab_mat_gpu,	sizeUintL,hipMemcpyDeviceToHost);
		if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to allocate copy array lab_mat_gpu from device to host (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
		sprintf(buffer,"/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/-4-lab_mat_cpu-root_equivalence.txt");
		write_labmat_tiled(lab_mat_cpu, tiledimY, tiledimX, ntilesX, ntilesY, buffer);
	}
	/* ....::: [2/3 stage] :::.... */




	/* ....::: [3/3 stage] INTRA-TILE #2 :::.... */

	intra_tile_re_label<<<grid,block,sh_mem>>>(NC,lab_mat_gpu);
	cudaLastErr 	= hipGetLastError();
	if (cudaLastErr != hipSuccess){ printf ("ERROR {intra_tile_re_label} -- %s\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	// D2H --> [lab_mat_cpu]
	//cudaLastErr = cudaMemcpyAsync( lab_mat_cpu,lab_mat_gpu,	sizeUintL,cudaMemcpyDeviceToHost, stream[0] );
	cudaLastErr 	= hipMemcpy( lab_mat_cpu,lab_mat_gpu,	sizeUintL,hipMemcpyDeviceToHost );
	if (cudaLastErr != hipSuccess){ fprintf(stderr, "Failed to copy array lab_mat_gpu from device to host lab_mat_cpu (error code %s)!\n", hipGetErrorString(cudaLastErr)); exit(EXIT_FAILURE); }
	/* INTERMEDIATE CHECK [activate/deactivate]*/
	if (printme){
		count_print++;
		printf("  -%d- %30s\n\n",count_print,"intra_tile_re_label");
		sprintf(buffer,"/home/giuliano/work/Projects/LIFE_Project/LUC_gpgpu/soil_sealing/data/-5-lab_mat_cpu-intra_tile_re_label.txt");
		write_labmat_tiled(lab_mat_cpu, tiledimY, tiledimX, ntilesX, ntilesY, buffer);
	}
	/* ....::: [3/3 stage] :::.... */

	end_t = clock();

	/* DO NOT EDIT THE FOLLOWING PRINT (it's used in MatLab to catch the elapsed time!)*/
	printf("Total time: %f [msec]\n", (double)(end_t - start_t) / CLOCKS_PER_SEC * 1000 );

	// SAVE lab_mat to file and compare with MatLab
	sprintf(buffer,Lcuda);
	write_labmat_matlab(lab_mat_cpu, tiledimX, tiledimY, ntilesX, ntilesY, buffer);

	// FREE MEMORY:
	hipHostFree(lab_mat_cpu);
	hipHostFree(urban_cpu);
	hipFree(lab_mat_gpu);
	hipFree(urban_gpu);
/*	cudaStreamDestroy( stream[0] );
	cudaStreamDestroy( stream[1] );
*/

	//printf("\nFinished!!\n");
	// RETURN:
	return 0;
}
